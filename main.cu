#include <stdio.h>
#include <hip/hip_runtime.h>
#include "sequential/render.h"
#include "cuda/render.h"



int main(int argc, char* argv[]){
    set_memory();
    set_memory_cuda();
  
    hipDeviceSynchronize();
    // render_sequential_barneshut(argc, argv);
    render_cuda_exhaustive(argc, argv);
    // render_cuda_exhaustive(argc, argv);

    hipDeviceSynchronize();
    
    free_memory();
    return 0;
}